#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <stdio.h>
#include <math.h>
#include <unistd.h>
#include "helpers.cuh"
#include <string.h>

#define DATASIZE 192000  //Define Data Size ( Sampling rate values )

#define EQ 10 //Define number of bands in equalizer

__global__  void equalize(hipfftComplex *data_FFT, hipfftReal *eq_D,int sample_inc,int BATCH);



char* itoa(int val, int base);
char *getFilename(int n,char* str);




int main (int argc,char **argv)
{

int BATCH=atoi(argv[1]);

//Input variables for equalizer,sample values

hipfftReal *hostInputData=(hipfftReal*)malloc(DATASIZE*BATCH*sizeof(hipfftReal));
hipfftReal *eq_strength=(hipfftReal*)malloc(EQ*BATCH*sizeof(hipfftReal));
  
//IO HANDLE ----------------------------------------------------------------- AUDIOIN  

FILE *file,*file_E;

int BATCH_NO=0; 

for(int b=0; b< BATCH; b++){
	
   char name_audio[15]="audioin"; 	
   char name_equalize[15]="equalizer";
   
    	
   char* file_name=getFilename(b,name_audio);	
   char* equ_file=getFilename(b,name_equalize); 	

   
   if( access( file_name, F_OK ) != -1 ) {
   file = fopen(file_name,"r");
   for(int i=0;i<DATASIZE;i++){
        fscanf(file,"%f ",&hostInputData[i + DATASIZE*BATCH_NO]);
   }  
   fclose(file);
 
   }else {
   printf("ERROR : NO SUCH FILE");
   return 0;
   }
   
   if( access( equ_file, F_OK ) != -1 ) {
   file_E = fopen(equ_file,"r");
   for(int i=0;i<EQ;i++){
        fscanf(file_E,"%f ",&eq_strength[i + EQ*BATCH_NO]);
   }  
   fclose(file_E);
   }else {
   printf("ERROR : NO SUCH FILE");
   return 0;
   }
   
   BATCH_NO++ ;
}


printf("%d \n ",BATCH);

//-----------------
	hipEvent_t start,stop;
	float elapsedtime;
	
	//the moment at which we start measuring the time
	hipEventCreate(&start);
	hipEventRecord(start,0);	
//---------------------------------------------------------


    // --- Device side input data allocation and initialization
    hipfftReal *deviceInputData; hipMalloc((void**)&deviceInputData, DATASIZE * BATCH * sizeof(hipfftReal));
    hipfftReal *equ_D; hipMalloc((void**)&equ_D, EQ * BATCH * sizeof(hipfftReal));
    
    hipMemcpy(deviceInputData, hostInputData, DATASIZE * BATCH * sizeof(hipfftReal), hipMemcpyHostToDevice);
    hipMemcpy(equ_D, eq_strength, EQ * BATCH * sizeof(hipfftReal), hipMemcpyHostToDevice);

    // --- Host side output data allocation
    //hipfftComplex *hostOutputData = (hipfftComplex*)malloc((DATASIZE / 2 + 1) * BATCH * sizeof(hipfftComplex));

    // --- Device side output data allocation
    hipfftComplex *deviceOutputData; hipMalloc((void**)&deviceOutputData, (DATASIZE / 2 + 1) * BATCH * sizeof(hipfftComplex));

    // --- Batched 1D FFTs
    hipfftHandle handle;
    int rank = 1;                           // --- 1D FFTs
    int n[] = { DATASIZE };                 // --- Size of the Fourier transform
    int istride = 1, ostride = 1;           // --- Distance between two successive input/output elements
    int idist = DATASIZE, odist = (DATASIZE / 2 + 1); // --- Distance between batches
    int inembed[] = { 0 };                  // --- Input size with pitch (ignored for 1D transforms)
    int onembed[] = { 0 };                  // --- Output size with pitch (ignored for 1D transforms)
    int batch = BATCH;                      // --- Number of batched executions
    hipfftPlanMany(&handle, rank, n, 
                  inembed, istride, idist,
                  onembed, ostride, odist, HIPFFT_R2C, batch);

    //hipfftPlan1d(&handle, DATASIZE, HIPFFT_R2C, BATCH);
    hipfftExecR2C(handle,  deviceInputData, deviceOutputData);


//---------------------
    hipFree(deviceInputData);
    free(hostInputData);
    
    

//EQUALIZE---------------------------------------------------------------------

dim3 block(16,16);
int threads=(int)ceil(pow((DATASIZE*BATCH)/256.0,1/3.0));

dim3 grid(threads,threads,threads);
equalize<<<grid,block>>>(deviceOutputData,equ_D,ceil((DATASIZE/2 + 1) /(float)EQ),BATCH);   checkCudaError();


//INVERSE TRANSFORM--------------------------------------
    hipfftHandle inverse;
    if (hipfftPlanMany(&inverse, rank,n, 
                  onembed, ostride, odist,
                  inembed, istride, idist,HIPFFT_C2R, batch)
     != HIPFFT_SUCCESS){ 
		fprintf(stderr, "CUFFT error: Plan creation failed\n");
		return 0;
	}
	
	hipfftReal *inversefft; 
	hipMalloc((void**)&inversefft, DATASIZE * BATCH * sizeof(hipfftReal));   checkCudaError();
  
	if (hipfftExecC2R(inverse,deviceOutputData,inversefft) != HIPFFT_SUCCESS){
		fprintf(stderr, "CUFFT error: ExecR2C Forward failed\n");	
		return 0;
	}
 
    hipFree(deviceOutputData);  checkCudaError();
    hipfftDestroy(handle);
    hipfftDestroy(inverse);
    
 
    // --- Device->Host copy of the results
    hipfftReal *opt = (hipfftReal*)malloc(DATASIZE* BATCH * sizeof(hipfftReal));
    
    if(opt == NULL){
	   printf("malloc() failed..!");
	   return 0;
	}
    
    hipMemcpy(opt,inversefft,DATASIZE * (BATCH) * sizeof(hipfftReal), hipMemcpyDeviceToHost);  checkCudaError();



//-----------------write to files

/*
FILE *file_Out;
BATCH_NO=0;

for(int i=0 ;i< BATCH ;i++){
 
    char name_audio_out[15]="audioi_out"; 
    char* file_name_2=getFilename(i,name_audio_out);	
     
    file_Out = fopen(file_name_2,"w");
    for (int j=BATCH_NO*DATASIZE;j<BATCH_NO*DATASIZE+DATASIZE;j++){
        fprintf(file_Out,"%f ",opt[j]/DATASIZE);
    }
    fclose(file_Out);
    BATCH_NO++; 
}
* 
*/

    hipfftDestroy(handle);


    
 //--------------------------------
 //the moment at which we stop measuring time 
	hipEventCreate(&stop);
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	
	//Find and print the elapsed time
	hipEventElapsedTime(&elapsedtime,start,stop);
	printf("Time spent for operation is %.10f seconds\n",elapsedtime/(float)1000);
	//we get the elapsedtime in milli seconds. Thats why we divide by 1000   
    

}

__global__  void equalize(hipfftComplex *data_FFT, hipfftReal *eq_D,int sample_inc,int BATCH){


   
int blockId = blockIdx.x+ blockIdx.y * gridDim.x
+ gridDim.x * gridDim.y * blockIdx.z;
int threadId = blockId * (blockDim.x * blockDim.y)
+ (threadIdx.y * blockDim.x) + threadIdx.x;

  
   if(  (0 <= threadId ) && (threadId < (DATASIZE/2 + 1)*BATCH)){
        int batch_id=(threadId/(DATASIZE/2 + 1));
        int rel_id=(threadId%(DATASIZE/2 + 1));
        //handle +1 case
        if(rel_id == (DATASIZE/2 + 1) -1 ){
		   	rel_id--;
		}
        int eq_id=(EQ*batch_id) + (rel_id / sample_inc);
        

        
        data_FFT[threadId].x=eq_D[eq_id]*data_FFT[threadId].x;
        data_FFT[threadId].y=eq_D[eq_id]*data_FFT[threadId].y;
   }
   
   //printf("IN"); 
    	
}

char* itoa(int val, int base){
	if(val == 0){
	   return "0";	
	}
	static char buf[32] = {0};
	int i = 30;
	for(; val && i ; --i, val /= base)
	  buf[i] = "0123456789abcdef"[val % base];
	return &buf[i+1];	
}


char* getFilename(int n,char* str){
   char* val=itoa(n,10);
   strcat(str,val);
   char *ext=".txt";
   strcat(str,ext);

   return str;
}


